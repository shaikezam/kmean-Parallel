#include "hip/hip_runtime.h"
#define _GNU_SOURCE
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h> 
#include <string.h>
#include <omp.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <conio.h>

typedef struct
{
	float* coordinates;
} Point;

typedef struct
{
	Point center;
	Point* points;
	float radius;
	int numOfPoints;
} Cluster;

__global__ void calculateCenter(float* coordinates, const int NUM_OF_POINTS)
{
    int i = threadIdx.x;
	//printf("Before: %f\n", coordinates[i]);
    coordinates[i] = coordinates[i] / NUM_OF_POINTS;
	//printf("After: %f\n", coordinates[i]);
}

__global__ void calculateCenter2(Point* points, float* coordinates)
{
    int i = threadIdx.x;
	int y = blockIdx.x;
	//printf("y = %d\n", y);
	//printf("i = %d\n", i);
	if(i == 0)
	{
		printf("index = %d\n",i);
	}
	else
	{
		printf("index = %d\n",(y+i)*i);
	}
	Point point = points[y];
	//printf("Before: %f\n", coordinates[i]);
    coordinates[i*y + (i - y)] = point.coordinates[i];
	//printf("After: %f\n", coordinates[i]);
}

__global__ void calculateDistance(float* coordinates1, float* coordinates2, double* sum)
{
    int i = threadIdx.x;
	//printf("Before: %f\n", coordinates[i]);
    double temp = (coordinates1[i] - coordinates2[i])*(coordinates1[i] - coordinates2[i]);
	//atomicAdd(&sum[0], temp);
	//printf("After: %f\n", coordinates[i]);
}

__device__ int counter;

__global__ void calculateClusterDGlobal(float* distances, float* cords, const int NUM_OF_DIMENSIONS, int NUM_OF_POINTS, int *current_thread_count)
{
	printf("%d\n", blockIdx.y);
	
	int mainPointIndex = blockIdx.x;
	int secondaryPointIndex = threadIdx.x;
	printf("%d\n", current_thread_count[0]);	
	int current_val = *current_thread_count;
	for (int j = 0 ; j < NUM_OF_DIMENSIONS ; j++)
	{
		float c1 = cords[mainPointIndex*NUM_OF_DIMENSIONS + j];
		float c2 = cords[secondaryPointIndex*NUM_OF_DIMENSIONS + j];
		distances[current_val] = (c1 - c2)*(c1 - c2);
	}
	
	distances[current_val] = sqrt(distances[current_val]);
	/*printf("againstPoint*NUM_OF_DIMENSIONS = %d\n", whichPoint*NUM_OF_DIMENSIONS);
	printf("againstPoint*NUM_OF_DIMENSIONS = %d\n", againstPoint*NUM_OF_DIMENSIONS);*/
	printf("distances[%d] = %f\n", current_val, distances[current_val]);
	atomicAdd(current_thread_count, 1);
}

/*int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}*/

// Helper function for using CUDA to add vectors in parallel.
float* calculateCenterUsingCuda(float* coordinates, const int NUM_OF_DIMENSIONS, const int NUM_OF_POINTS)
{
    float* coordinates_dev = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    cudaStatus = hipMalloc((void**)&coordinates_dev, NUM_OF_DIMENSIONS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMemcpy(coordinates_dev, coordinates, NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    calculateCenter <<< 1, NUM_OF_DIMENSIONS >>>(coordinates_dev, NUM_OF_POINTS);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "calculateCenterlaunch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateCenter!\n", cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(coordinates, coordinates_dev, NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	hipFree(coordinates_dev);

	cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

    return coordinates;
}

float* calculateCenterUsingCuda2(Cluster* cluster, const int NUM_OF_DIMENSIONS)
{
	const int numOfPoints = cluster->numOfPoints;
    float* coordinates_dev;
	float* coordinates = (float*)calloc(NUM_OF_DIMENSIONS * numOfPoints, sizeof(float));
	Point* points_dev;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    cudaStatus = hipMalloc((void**)&coordinates_dev, NUM_OF_DIMENSIONS * numOfPoints * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMalloc((void**)&points_dev, numOfPoints * sizeof(Point));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMemcpy(coordinates_dev, coordinates, NUM_OF_DIMENSIONS * numOfPoints * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	cudaStatus = hipMemcpy(points_dev, cluster->points, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    calculateCenter2 <<<numOfPoints, NUM_OF_DIMENSIONS>>>(points_dev, coordinates_dev);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "calculateCenterlaunch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateCenter!\n", cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(coordinates, coordinates_dev, NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	hipFree(coordinates_dev);
	hipFree(points_dev);

	cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

    return coordinates;
}

double* calculateDistanceBetween2PointsUsingCuda(float* coordinates1, float* coordinates2, double* sum, const int NUM_OF_DIMENSIONS)
{
	double* sum_dev = 0;
    float* coordinates_dev1 = 0;
	float* coordinates_dev2 = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    cudaStatus = hipMalloc((void**)&coordinates_dev1, NUM_OF_DIMENSIONS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMalloc((void**)&coordinates_dev2, NUM_OF_DIMENSIONS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMalloc((void**)&sum_dev, 1 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMemcpy(coordinates_dev1, coordinates1, NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	cudaStatus = hipMemcpy(coordinates_dev2, coordinates2, NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	cudaStatus = hipMemcpy(sum_dev, sum, 1 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    calculateDistance <<< 1, NUM_OF_DIMENSIONS >>>(coordinates_dev1, coordinates_dev2, sum);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "calculateCenterlaunch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateCenter!\n", cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(coordinates1, coordinates_dev1, NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	cudaStatus = hipMemcpy(coordinates2, coordinates_dev2, NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	cudaStatus = hipMemcpy(sum, sum_dev, 1 * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	hipFree(coordinates_dev1);
	hipFree(coordinates_dev2);
	hipFree(sum_dev);

	cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

    return sum;
}

float* calculateClusterD(Cluster* cluster, const int NUM_OF_DIMENSIONS)
{
	int tally = 0, *dev_tally;

	hipMalloc((void **)&dev_tally, sizeof(int));

	hipMemcpy(dev_tally, &tally, sizeof(int), hipMemcpyHostToDevice);

	float* temp_cords = (float*)calloc(cluster->numOfPoints  * NUM_OF_DIMENSIONS, sizeof(float));
	float* temp_cords_dev;
	int count = 0;
	for(int i = 0 ; i < cluster->numOfPoints ; i++)
	{
		for(int j = 0 ; j < NUM_OF_DIMENSIONS ; j++)
		{
			temp_cords[count] = cluster->points[i].coordinates[j];
			count++;
		}
	}
	float* temp_distances = (float*)calloc(cluster->numOfPoints  * cluster->numOfPoints * NUM_OF_DIMENSIONS, sizeof(float));
	float* temp_distances_dev;
	hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

	cudaStatus = hipMalloc((void**)&temp_cords_dev, cluster->numOfPoints  * NUM_OF_DIMENSIONS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMalloc((void**)&temp_distances_dev, cluster->numOfPoints  * cluster->numOfPoints * NUM_OF_DIMENSIONS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

	cudaStatus = hipMemcpy(temp_cords_dev, temp_cords, cluster->numOfPoints  * NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	cudaStatus = hipMemcpy(temp_distances_dev, temp_distances, cluster->numOfPoints * cluster->numOfPoints * NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	calculateClusterDGlobal <<< cluster->numOfPoints, cluster->numOfPoints >>>(temp_distances_dev, temp_cords_dev, NUM_OF_DIMENSIONS, cluster->numOfPoints, dev_tally);

	cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "calculateCenterlaunch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateCenter!\n", cudaStatus);
    }

	// Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(temp_distances, temp_distances_dev, cluster->numOfPoints * cluster->numOfPoints * NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

	hipMemcpy(&tally, dev_tally, sizeof(int), hipMemcpyDeviceToHost); 
	printf("total number of threads that executed was: %d\n", tally);

	//hipFree(temp_distances_dev);
	//hipFree(temp_cords_dev);
	//free(temp_cords);
	return temp_distances;
}

/*hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}*/
