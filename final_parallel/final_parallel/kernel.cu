#include "hip/hip_runtime.h"
#define _GNU_SOURCE
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h> 
#include <string.h>
#include <omp.h>
#include <iostream>
#include <conio.h>

typedef struct
{
	float* coordinates;
} Point;

typedef struct
{
	Point center;
	Point* points;
	float radius;
	int numOfPoints;
} Cluster;

__global__ void calculateCenter(Point* point, const int NUM_OF_POINTS)
{
    int i = threadIdx.x;
    point->coordinates[i] = point->coordinates[i] / NUM_OF_POINTS;
}

/*int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}*/

// Helper function for using CUDA to add vectors in parallel.
Point* calculateCenterUsingCuda(Point* point, const int NUM_OF_DIMENSIONS, const int NUM_OF_POINTS)
{
     Point* point_dev;

    Point point_for_dev;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    cudaStatus = hipMalloc((void**)&point_dev, 1 * sizeof(Point));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMalloc((void**)&(point_for_dev.coordinates), NUM_OF_DIMENSIONS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMemcpy(point_dev, &point_for_dev, 1 * sizeof(Point), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus = hipMemcpy(point_for_dev.coordinates, point->coordinates, NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    calculateCenter <<< 1, NUM_OF_DIMENSIONS >>>(point_dev, NUM_OF_POINTS);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "calculateCenterlaunch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateCenter!\n", cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&point_for_dev , point_dev, 1 * sizeof(Point), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus = hipMemcpy(point->coordinates , point_for_dev.coordinates, NUM_OF_DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

    return point;
}

/*hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}*/
